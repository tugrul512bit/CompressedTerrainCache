#include "hip/hip_runtime.h"
#include "CompressedTerrainCache.cuh"
namespace CompressedTerrainCache {
	namespace Kernels {
		// Each block decodes a tile concurrently with each block thread decoding its own column in a striped-pattern.

		__global__ void k_decodeTile(
			unsigned char* encodedTiles,
			unsigned char* encodedTrees,
			uint32_t blockAlignedElements,
			uint32_t tileSizeBytes,
			unsigned char* originalTileDataForComparison,
			uint32_t numTilesToTest,
			uint32_t terrainWidth,
			uint32_t terrainHeight,
			uint32_t tileWidth,
			uint32_t tileHeight) {
			uint32_t numTilesX = (terrainWidth + tileWidth + 1) / tileWidth;
			uint32_t numTilesY = (terrainHeight + tileHeight + 1) / tileHeight;
			uint32_t localThreadIndex = threadIdx.x;
			uint32_t numBlocks = gridDim.x;
			uint32_t globalThreadIndex = localThreadIndex + blockIdx.x * HuffmanTileEncoder::NUM_CUDA_THREADS_PER_BLOCK;
			uint32_t numGlobalThreads = HuffmanTileEncoder::NUM_CUDA_THREADS_PER_BLOCK * numBlocks;
			uint32_t* treePtr = reinterpret_cast<uint32_t*>(encodedTrees);
			uint32_t* tilePtr = reinterpret_cast<uint32_t*>(encodedTiles);
			uint32_t blockAlignedBytes = blockAlignedElements * sizeof(uint32_t);
			// Tile steps.
			uint32_t numTileSteps = (numTilesToTest + numBlocks - 1) / numBlocks;
			for (uint32_t tileStep = 0; tileStep < numTileSteps; tileStep++) {
				uint32_t tile = tileStep * numBlocks + blockIdx.x;
				if (tile < numTilesToTest) {
					// Decode steps.
					uint32_t numDecodeSteps = (blockAlignedBytes + HuffmanTileEncoder::NUM_CUDA_THREADS_PER_BLOCK - 1) / HuffmanTileEncoder::NUM_CUDA_THREADS_PER_BLOCK;
					uint32_t decodeBitIndex = 0;
					uint32_t one = 1; 
					uint32_t* chunkBlockPtr = &tilePtr[blockAlignedElements * tile];
					uint32_t* treeBlockPtr = &treePtr[512 * tile]; 
					for (uint32_t decodeStep = 0; decodeStep < numDecodeSteps; decodeStep++) {

						uint32_t byteIndex = decodeStep * HuffmanTileEncoder::NUM_CUDA_THREADS_PER_BLOCK + localThreadIndex;
						if (byteIndex < blockAlignedBytes) {
							bool leafNodeFound = false;
							uint32_t currentNodeIndex = 0;
							uint8_t symbol = 0; 
							while (!leafNodeFound) {
								uint32_t chunkColumn = localThreadIndex;
								uint32_t chunkRow = decodeBitIndex / 32;
								uint32_t chunkBit = decodeBitIndex % 32;
								uint32_t chunk = chunkBlockPtr[chunkColumn + chunkRow * HuffmanTileEncoder::NUM_CUDA_THREADS_PER_BLOCK];
								uint32_t bitBeingDecoded = (chunk >> chunkBit) & one;
								uint32_t node = treeBlockPtr[1 + currentNodeIndex];
								uint8_t leafNode = node >> 8;
								uint16_t childNodeStart = node >> 16;
								if (!leafNode) {
									if (bitBeingDecoded) {
										currentNodeIndex = childNodeStart + 1;
									}	else {
										currentNodeIndex = childNodeStart;
									}
								}
								else {
									symbol = node & 0b11111111;
								}
								decodeBitIndex++;
								leafNodeFound = leafNode;
							}
							if (tileStep == 0) printf("%c", symbol);
							if(tileStep == 0)
							{
								uint32_t blockRow = tile / numTilesX;
								uint32_t blockColumn = tile % numTilesX;
								uint32_t chunkRow = decodeBitIndex / 32;
								uint32_t chunkBit = decodeBitIndex % 32;
								uint32_t col = blockColumn * tileWidth + (byteIndex % tileWidth);
								uint32_t row = blockRow * tileHeight + (byteIndex / tileWidth);
								if (byteIndex < tileSizeBytes && col < terrainWidth && row < terrainHeight) {
									if (symbol != originalTileDataForComparison[tileSizeBytes * tile + byteIndex]) {
										printf("\nERROR! Encoded data - original data mismatch = %u %u. \n", col, row);
									}
								}
							}
						}
					}
				}
			}
		}
	}
}