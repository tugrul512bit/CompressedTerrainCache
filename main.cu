#include "hip/hip_runtime.h"
﻿#include <stdio.h>
#include <random>
#include <cmath>
// Cached encoder tool for 2d terrain.
#include "CompressedTerrainCache.cuh"
// OpenCV4 for 2d render.
#include<opencv2/opencv.hpp>

int main()
{
    // Player can see this far (in units).
    uint64_t playerVisibilityRadius = 2000;
    // Low velocity is more cache-friendly, high velocity causes more decoding and PCIE utilization.
    float playerOrbitAngularVelocity = 0.005f;
    // 2D terrain map size (in units), 2.5GB for terrain data, no allocation on device memory.
    uint64_t terrainWidth = 15001;
    uint64_t terrainHeight = 15003;
    // 2D tile size (in units). tileWidth * tileHeight = (multiple of HuffmanTileEncoder::NUM_CUDA_THREADS_PER_BLOCK) is preferred for performing better.
    uint64_t tileWidth = 64;
    uint64_t tileHeight = 64;
    // Tile cache size, in tiles (so that 250x250 cache can store 62500 tiles at once with 1.2GB device memory allocation for 64x64 tile size)
    uint64_t tileCacheSlotColumns = 70;
    uint64_t tileCacheSlotRows = 70;
    // internally this calculation is used as ordering of tiles.(index = tileX + tileY * numTilesX) (row-major)
    uint64_t numTerrainElements = terrainWidth * terrainHeight;
    uint64_t numTilesX = (terrainWidth + tileWidth - 1) / tileWidth;
    uint64_t numTilesY = (terrainHeight + tileHeight - 1) / tileHeight;
    uint64_t numTiles = numTilesX * numTilesY;
    // Uses 2x memory, 1 for slow method, 1 for fast method.
    bool benchmarkSlowMethodForComparison = true;


    // Terrain element type (only POD structs/types are allowed). Uncomment below to select different sized terrain elements (example rendering will adapt colors automatically).
    //using T = unsigned char;
    using T = uint32_t;
    //using T = uint64_t;

    // Generating sample terrain (2D cos wave pattern).
    std::shared_ptr<T> terrain = std::shared_ptr<T>(new T[numTerrainElements], [](T* ptr) { delete[] ptr; });
    uint32_t colorScale = (sizeof(T) == 8 ? 255 : 1);

    for (uint64_t y = 0; y < terrainHeight; y++) {
        for (uint64_t x = 0; x < terrainWidth; x++) {
            uint64_t index = x + y * terrainWidth;

            uint32_t blue = (77 + cos(x * 0.002f) * cos(y * 0.002f) * 50) * colorScale;
            uint32_t green = (37 + cos(x * 0.0005f) * cos(y * 0.0005f) * 20) * colorScale;
            uint32_t red = (130 + cos(x * 0.0004f) * cos(y * 0.0004f) * 100) * colorScale;
            uint32_t alpha = 255 * colorScale;
            terrain.get()[index] = ((sizeof(T) == 8) ? (blue | (green << 16) | (red << 32) | (0xFFFF << 48)) : ((sizeof(T) == 4) ? (blue | (green << 8) | (red << 16) | (0xFF << 24)) : blue));
        }
    }

    // Creating tile manager that uses terrain as input.
    int deviceIndex = 0; // 0 means first cuda gpu, 1 means second cuda gpu, ...
    int numCpuThreads = std::thread::hardware_concurrency();
    CompressedTerrainCache::TileManager<T> tileManager(terrain.get(), terrainWidth, terrainHeight, tileWidth, tileHeight, tileCacheSlotColumns, tileCacheSlotRows,  numCpuThreads, deviceIndex);
    
    // Rendering reference terrain in a window.
    cv::namedWindow("Downscaled Raw Terrain Data");
    cv::resizeWindow("Downscaled Raw Terrain Data", 1024, 1024);
    cv::Mat img(terrainHeight, terrainWidth, sizeof(T) == 4 ? CV_8UC4 : (sizeof(T) == 8 ? CV_16UC4 : CV_8UC1), terrain.get());
    cv::Mat downScaledImg;
    cv::resize(img, downScaledImg, cv::Size(1024, 1024), 0, 0, cv::INTER_AREA);
    cv::imshow("Downscaled Raw Terrain Data", downScaledImg);
    cv::waitKey(1);
    cv::namedWindow("Loaded Tiles");
    cv::resizeWindow("Loaded Tiles", 1024, 1024);

    float angle = 0.0f;
    double timeNormalAccess = 0.0f;
    double timeDecode = 0.0f;
    double dataSizeNormalAccess = 0.0f;
    double dataSizeDecode = 0.0f;
    double throughputNormalAccess = 0.0f;
    double throughputDecode = 0.0;
    unsigned char* loadedTilesOnDevice_d = nullptr;
    constexpr int ACCESS_METHOD_DIRECT = 0;
    constexpr int ACCESS_METHOD_DECODE_HUFFMAN_CACHED = 1;
    int accessMethod = ACCESS_METHOD_DECODE_HUFFMAN_CACHED;
    // Sample game loop.
    while (true) {
        angle += playerOrbitAngularVelocity;
        // Creating a sample list of tile-indices using visibility range of player.
        std::vector<uint32_t> tileIndexList;
        for (uint64_t tileY = 0; tileY < numTilesY; tileY++) {
            for (uint64_t tileX = 0; tileX < numTilesX; tileX++) {
                // Checking if player visibility range collides with the current tile.
                uint64_t playerX = terrainWidth / 2 + cos(angle) * terrainWidth / 4;
                uint64_t playerY = terrainHeight / 2 + sin(angle) * terrainHeight / 4;
                uint64_t distanceX = playerX - (tileX * tileWidth + tileWidth / 2);
                uint64_t distanceY = playerY - (tileY * tileHeight + tileHeight / 2);
                uint64_t distance = sqrt(distanceX * distanceX + distanceY * distanceY);
                if (distance < playerVisibilityRadius) {
                    tileIndexList.push_back(tileX + tileY * numTilesX);
                }
            }
        }

        if (benchmarkSlowMethodForComparison) {
            accessMethod = 1 - accessMethod;
        }
        switch (accessMethod) {
            case ACCESS_METHOD_DIRECT: loadedTilesOnDevice_d = tileManager.accessSelectedTiles(tileIndexList, &timeNormalAccess, &dataSizeNormalAccess, &throughputNormalAccess); break;
            case ACCESS_METHOD_DECODE_HUFFMAN_CACHED:loadedTilesOnDevice_d = tileManager.decodeSelectedTiles(tileIndexList, &timeDecode, &dataSizeDecode, &throughputDecode); break;
            default:break;
        }
       
        uint64_t outputBytes = tileIndexList.size() * (size_t)tileWidth * tileHeight * sizeof(T);
        std::vector<T> loadedTilesOnHost_h(tileIndexList.size() * (size_t)tileWidth * tileHeight);
        // Downloading output tile data from device memory to RAM.
        CUDA_CHECK(hipMemcpy(loadedTilesOnHost_h.data(), loadedTilesOnDevice_d, outputBytes, hipMemcpyDeviceToHost));
        // Clearing old terrain to see if visibility range works correctly.
        std::fill(terrain.get(), terrain.get() + (terrainWidth * terrainHeight), sizeof(T) == 1 ? 255 : 0);
        uint32_t num = tileIndexList.size();

        for (uint32_t i = 0; i < num; i++) {
            uint32_t tileIndex = tileIndexList[i];
            uint32_t tileX = tileIndex % numTilesX;
            uint32_t tileY = tileIndex / numTilesX;
            for (uint32_t y = 0; y < tileHeight; y++) {
                for (uint32_t x = 0; x < tileWidth; x++) {
                    uint64_t terrainX = (tileX * tileWidth + x);
                    uint64_t terrainY = (tileY * tileHeight + y);
                    uint64_t terrainDestinationIndex = terrainX + terrainY * (uint64_t)terrainWidth;
                    uint64_t sourceIndex = i * (uint64_t)tileWidth * tileHeight + x + y * tileWidth;
                    if (terrainX < terrainWidth && terrainY < terrainHeight) {
                        terrain.get()[terrainDestinationIndex] = loadedTilesOnHost_h[sourceIndex];
                    }
                }
            }
        }
        // Rendering benchmark window.
        cv::Mat img2(terrainHeight, terrainWidth, sizeof(T) == 4 ? CV_8UC4 : (sizeof(T) == 8 ? CV_16UC4 : CV_8UC1), terrain.get());
        cv::Mat downScaledImg2;
        cv::resize(img2, downScaledImg2, cv::Size(1024, 1024), 0, 0, cv::INTER_AREA);
        std::string directMethod = std::string("Unified memory tile stream:");
        std::string decodeInfo1 = std::string("Kernel = ") + std::to_string(timeNormalAccess) + std::string(" seconds");
        std::string decodeInfo2 = std::string("Data = ") + std::to_string(dataSizeNormalAccess) + std::string(" GB");
        std::string decodeInfo3 = std::string("Throughput = ") + std::to_string(throughputNormalAccess) + std::string(" GB/s");
        std::string decodeMethod = std::string("Unified memory encoded-tile stream + decoding + 2D caching:");
        std::string decodeInfo4 = std::string("Kernel = ") + std::to_string(timeDecode) + std::string(" seconds");
        std::string decodeInfo5 = std::string("Data = ") + std::to_string(dataSizeDecode) + std::string(" GB");
        std::string decodeInfo6 = std::string("Throughput = ") + std::to_string(throughputDecode) + std::string(" GB/s");
        cv::Mat benchmark;
        auto color1 = cv::Scalar(0, 255 * colorScale, 255 * colorScale, 255 * colorScale);
        auto color2 = cv::Scalar(0, 255 * colorScale, 0, 255 * colorScale);
        auto color3 = cv::Scalar(255 * colorScale, 255 * colorScale, 0, 255 * colorScale);
        cv::putText(downScaledImg2, directMethod, cv::Point(20, 60), cv::FONT_HERSHEY_SIMPLEX, 0.75, color1, 2, cv::LINE_AA);
        cv::putText(downScaledImg2, decodeInfo1, cv::Point(20, 80), cv::FONT_HERSHEY_SIMPLEX, 0.75, color1, 2, cv::LINE_AA);
        cv::putText(downScaledImg2, decodeInfo2, cv::Point(20, 100), cv::FONT_HERSHEY_SIMPLEX, 0.75, color1, 2, cv::LINE_AA);
        cv::putText(downScaledImg2, decodeInfo3, cv::Point(20, 120), cv::FONT_HERSHEY_SIMPLEX, 0.75, color1, 2, cv::LINE_AA);
        cv::putText(downScaledImg2, decodeMethod, cv::Point(20, 160), cv::FONT_HERSHEY_SIMPLEX, 0.75, color2, 2, cv::LINE_AA);
        cv::putText(downScaledImg2, decodeInfo4, cv::Point(20, 180), cv::FONT_HERSHEY_SIMPLEX, 0.75, color2, 2, cv::LINE_AA);
        cv::putText(downScaledImg2, decodeInfo5, cv::Point(20, 200), cv::FONT_HERSHEY_SIMPLEX, 0.75, color2, 2, cv::LINE_AA);
        cv::putText(downScaledImg2, decodeInfo6, cv::Point(20, 220), cv::FONT_HERSHEY_SIMPLEX, 0.75, color2, 2, cv::LINE_AA);
        cv::putText(downScaledImg2, "Press ESC to exit", cv::Point(20, 980), cv::FONT_HERSHEY_SIMPLEX, 0.75, color3, 2, cv::LINE_AA);
        cv::imshow("Loaded Tiles", downScaledImg2);
        int key = cv::waitKey(1);
        if (key == 27) {
            break;
        }
    }
    cv::destroyAllWindows();
    return 0;
}