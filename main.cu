#include "hip/hip_runtime.h"
﻿#include <stdio.h>
#include <random>
#include <cmath>
// Cached encoder tool for 2d terrain.
#include "CompressedTerrainCache.cuh"
// For OpenCV4 for 2d render and keeping track of which tiles are updated.
#include<opencv2/opencv.hpp>
#include <map>
// For handling terrain updates by mouse click
struct UserTerrainUpdate {
    int bytesPerTerrainElement;
    int terrainWidth;
    int terrainHeight;
    int tileWidth;
    int tileHeight;
    int numTilesX;
    unsigned char* terrain;
    cv::Mat img;
    cv::Mat downScaledImg;
    std::vector<int> updateTileList;
};
void clickEvent(int event, int mx, int my, int flags, void* userData)
{
    UserTerrainUpdate* terrain = reinterpret_cast<UserTerrainUpdate*>(userData);
    std::map<int, bool> indicesOfUpdatedTiles;
    if (cv::EVENT_LBUTTONDOWN == event) {
        int64_t mapX = (mx / 1024.0) * terrain->terrainWidth;
        int64_t mapY = (my / 1024.0) * terrain->terrainHeight;
        for (int64_t y = -100; y <= 100; y++) {
            for (int64_t x = -100; x <= 100; x++) {
                int64_t xt = x + mapX;
                int64_t yt = y + mapY;
                int64_t r = sqrt(x * x + y * y);
                if (r < 100) {
                    if (xt >= 0 && xt < terrain->terrainWidth && yt >= 0 && yt < terrain->terrainHeight) {
                        indicesOfUpdatedTiles[xt / terrain->tileWidth + (yt / terrain->tileHeight) * terrain->numTilesX] = true;
                        for (int64_t i = 0; i < terrain->bytesPerTerrainElement; i++) {
                            terrain->terrain[i + (xt + yt * terrain->terrainWidth) * terrain->bytesPerTerrainElement] = 255;
                        }
                    }
                }
            }
        }
        terrain->updateTileList.clear();
        for (auto e : indicesOfUpdatedTiles) {
            terrain->updateTileList.push_back(e.first);
        }
        cv::resize(terrain->img, terrain->downScaledImg, cv::Size(1024, 1024), 0, 0, cv::INTER_AREA);
        cv::imshow("Downscaled Raw Terrain Data. Click anywhere to edit the terrain.", terrain->downScaledImg);
    }
    else if (cv::EVENT_RBUTTONDOWN == event) {

    }
    else if (cv::EVENT_MBUTTONDOWN == event) {
        
    }
}
int main()
{
    // Player can see this far (in units).
    uint64_t playerVisibilityRadius = 2000;
    // Low velocity is more cache-friendly, high velocity causes more decoding and PCIE utilization.
    float playerOrbitAngularVelocity = 0.009f;
    // 2D terrain map size (in units), 2.5GB for terrain data, no allocation on device memory.
    uint64_t terrainWidth = 11001;
    uint64_t terrainHeight = 11003;
    // 2D tile size (in units).
    uint64_t tileWidth = 64;
    uint64_t tileHeight = 64;
    // Tile cache size, in tiles (so that 64x64 cache can store 4096 tiles at once). Consumes device memory.
    uint64_t tileCacheSlotColumns = 64;
    uint64_t tileCacheSlotRows = 64;
    // internally this calculation is used as ordering of tiles.(index = tileX + tileY * numTilesX) (row-major)
    uint64_t numTerrainElements = terrainWidth * terrainHeight;
    uint64_t numTilesX = (terrainWidth + tileWidth - 1) / tileWidth;
    uint64_t numTilesY = (terrainHeight + tileHeight - 1) / tileHeight;
    uint64_t numTiles = numTilesX * numTilesY;
    // Uses 2x memory, 1 for slow method, 1 for fast method. Slow method only demonstrates unoptimized access to terrain to compare to optimized version that uses decoding and caching.
    bool benchmarkSlowMethodForComparison = true;


    // Terrain element type (only POD structs/types are allowed). Uncomment below to select different sized terrain elements (example rendering will adapt colors automatically).
    //using T = unsigned char;
    using T = uint32_t;
    //using T = uint64_t;

    // Generating sample terrain (2D cos wave pattern).
    std::vector<T> terrain = std::vector<T>(numTerrainElements);
    std::vector<T> terrainBenchmark = std::vector<T>(numTerrainElements);
    uint32_t colorScale = (sizeof(T) == 8 ? 255 : 1);

    for (uint64_t y = 0; y < terrainHeight; y++) {
        for (uint64_t x = 0; x < terrainWidth; x++) {
            uint64_t index = x + y * terrainWidth;
            uint32_t blue = (77 + cos(x * 0.002f) * cos(y * 0.002f) * 50) * colorScale;
            uint32_t green = (37 + cos(x * 0.0005f) * cos(y * 0.0005f) * 20) * colorScale;
            uint32_t red = (130 + cos(x * 0.0004f) * cos(y * 0.0004f) * 100) * colorScale;
            uint32_t alpha = 255 * colorScale;
            terrain[index] = ((sizeof(T) == 8) ? (blue | (green << 16) | (red << 32) | (0xFFFF << 48)) : ((sizeof(T) == 4) ? (blue | (green << 8) | (red << 16) | (0xFF << 24)) : blue));
        }
    }

    // Creating tile manager that uses terrain as input.
    int deviceIndex = 0; // 0 means first cuda gpu, 1 means second cuda gpu, ...
    int numCpuThreads = std::thread::hardware_concurrency();
    std::cout << "Encoding tiles." << std::endl;
    CompressedTerrainCache::TileManager<T> tileManager(terrain.data(), terrainWidth, terrainHeight, tileWidth, tileHeight, tileCacheSlotColumns, tileCacheSlotRows,  numCpuThreads, deviceIndex);
    std::cout << "Creating output windows." << std::endl;
    // Rendering reference terrain in a window.
    cv::namedWindow("Downscaled Raw Terrain Data. Click anywhere to edit the terrain.");
    cv::resizeWindow("Downscaled Raw Terrain Data. Click anywhere to edit the terrain.", 1024, 1024);
    cv::Mat img(terrainHeight, terrainWidth, sizeof(T) == 4 ? CV_8UC4 : (sizeof(T) == 8 ? CV_16UC4 : CV_8UC1), terrain.data());
    cv::Mat downScaledImg;
    UserTerrainUpdate userUpdateEventObj;
    userUpdateEventObj.bytesPerTerrainElement = sizeof(T);
    userUpdateEventObj.terrainWidth = terrainWidth;
    userUpdateEventObj.terrainHeight = terrainHeight;
    userUpdateEventObj.tileWidth = tileWidth;
    userUpdateEventObj.tileHeight = tileHeight;
    userUpdateEventObj.numTilesX = numTilesX;
    userUpdateEventObj.terrain = reinterpret_cast<unsigned char*>(terrain.data());
    userUpdateEventObj.img = img;
    userUpdateEventObj.downScaledImg = downScaledImg;
    cv::setMouseCallback("Downscaled Raw Terrain Data. Click anywhere to edit the terrain.", clickEvent, &userUpdateEventObj);
    cv::resize(img, downScaledImg, cv::Size(1024, 1024), 0, 0, cv::INTER_AREA);
    cv::imshow("Downscaled Raw Terrain Data. Click anywhere to edit the terrain.", downScaledImg);
    cv::waitKey(1);
    cv::namedWindow("Loaded Tiles");
    cv::resizeWindow("Loaded Tiles", 1024, 1024);

    float angle = 0.0f;
    double timeNormalAccess = 0.0f;
    double timeDecode = 0.0f;
    double dataSizeNormalAccess = 0.0f;
    double dataSizeDecode = 0.0f;
    double throughputNormalAccess = 0.0f;
    double throughputDecode = 0.0;
    unsigned char* loadedTilesOnDevice_d = nullptr;
    constexpr int ACCESS_METHOD_DIRECT = 0;
    constexpr int ACCESS_METHOD_DECODE_HUFFMAN_CACHED = 1;
    int accessMethod = ACCESS_METHOD_DECODE_HUFFMAN_CACHED;

    // Preparing benchmark window.
    cv::Mat img2(terrainHeight, terrainWidth, sizeof(T) == 4 ? CV_8UC4 : (sizeof(T) == 8 ? CV_16UC4 : CV_8UC1), terrainBenchmark.data());
    cv::Mat downScaledImg2;
    // Sample game loop.
    while (true) {
        angle += playerOrbitAngularVelocity;
        // Check if user updated the original terrain, and re-encode if necessary.
        if (userUpdateEventObj.updateTileList.size() > 0) {
            std::cout << std::endl;
            tileManager.encodeTerrain();
            tileManager.invalidateCache();
            userUpdateEventObj.updateTileList.clear();
        }

        // Creating a sample list of tile-indices using visibility range of player.
        std::vector<uint32_t> tileIndexList;
        for (uint64_t tileY = 0; tileY < numTilesY; tileY++) {
            for (uint64_t tileX = 0; tileX < numTilesX; tileX++) {
                // Checking if player visibility range collides with the current tile.
                uint64_t playerX = terrainWidth / 2 + cos(angle) * terrainWidth / 4;
                uint64_t playerY = terrainHeight / 2 + sin(angle) * terrainHeight / 4;
                uint64_t distanceX = playerX - (tileX * tileWidth + tileWidth / 2);
                uint64_t distanceY = playerY - (tileY * tileHeight + tileHeight / 2);
                uint64_t distance = sqrt(distanceX * distanceX + distanceY * distanceY);
                if (distance < playerVisibilityRadius) {
                    tileIndexList.push_back(tileX + tileY * numTilesX);
                }
            }
        }

        if (benchmarkSlowMethodForComparison) {
            accessMethod = 1 - accessMethod;
        }
        switch (accessMethod) {
            case ACCESS_METHOD_DIRECT: loadedTilesOnDevice_d = tileManager.accessSelectedTiles(tileIndexList, &timeNormalAccess, &dataSizeNormalAccess, &throughputNormalAccess); break;
            case ACCESS_METHOD_DECODE_HUFFMAN_CACHED:loadedTilesOnDevice_d = tileManager.decodeSelectedTiles(tileIndexList, &timeDecode, &dataSizeDecode, &throughputDecode); break;
            default:break;
        }
       
        uint64_t outputBytes = tileIndexList.size() * (size_t)tileWidth * tileHeight * sizeof(T);
        std::vector<T> loadedTilesOnHost_h(tileIndexList.size() * (size_t)tileWidth * tileHeight);
        // Downloading output tile data from device memory to RAM.
        CUDA_CHECK(hipMemcpy(loadedTilesOnHost_h.data(), loadedTilesOnDevice_d, outputBytes, hipMemcpyDeviceToHost));
        // Clearing old terrain to see if visibility range works correctly.
        uint32_t num = tileIndexList.size();
        uint64_t numErrors = 0;
        std::fill(terrainBenchmark.begin(), terrainBenchmark.end(), 0);
        for (uint32_t i = 0; i < num; i++) {
            uint32_t tileIndex = tileIndexList[i];
            uint32_t tileX = tileIndex % numTilesX;
            uint32_t tileY = tileIndex / numTilesX;
            for (uint32_t y = 0; y < tileHeight; y++) {
                for (uint32_t x = 0; x < tileWidth; x++) {
                    uint64_t terrainX = (tileX * tileWidth + x);
                    uint64_t terrainY = (tileY * tileHeight + y);
                    uint64_t terrainDestinationIndex = terrainX + terrainY * (uint64_t)terrainWidth;
                    uint64_t sourceIndex = i * (uint64_t)tileWidth * tileHeight + x + y * tileWidth;
                    if (terrainX < terrainWidth && terrainY < terrainHeight) {
                        terrainBenchmark[terrainDestinationIndex] = loadedTilesOnHost_h[sourceIndex];
                        numErrors += (terrain[terrainDestinationIndex] != terrainBenchmark[terrainDestinationIndex]);
                    }
                }
            }
        }
        if (numErrors > 0) {
            std::cout << numErrors << " errors detected" << std::endl;
        }

        // Rendering benchmark window.
        cv::resize(img2, downScaledImg2, cv::Size(1024, 1024), 0, 0, cv::INTER_AREA);
        std::string directMethod = std::string("Unified memory tile stream:");
        std::string decodeInfo1 = std::string("Kernel = ") + std::to_string(timeNormalAccess) + std::string(" seconds");
        std::string decodeInfo2 = std::string("Data = ") + std::to_string(dataSizeNormalAccess) + std::string(" GB");
        std::string decodeInfo3 = std::string("Throughput = ") + std::to_string(throughputNormalAccess) + std::string(" GB/s");
        std::string decodeMethod = std::string("Unified memory encoded-tile stream + decoding + 2D caching:");
        std::string decodeInfo4 = std::string("Kernel = ") + std::to_string(timeDecode) + std::string(" seconds");
        std::string decodeInfo5 = std::string("Data = ") + std::to_string(dataSizeDecode) + std::string(" GB");
        std::string decodeInfo6 = std::string("Throughput = ") + std::to_string(throughputDecode) + std::string(" GB/s");
        cv::Mat benchmark;
        auto color1 = cv::Scalar(0, 255 * colorScale, 255 * colorScale, 255 * colorScale);
        auto color2 = cv::Scalar(0, 255 * colorScale, 0, 255 * colorScale);
        auto color3 = cv::Scalar(255 * colorScale, 255 * colorScale, 0, 255 * colorScale);
        cv::putText(downScaledImg2, directMethod, cv::Point(20, 60), cv::FONT_HERSHEY_SIMPLEX, 0.75, color1, 2, cv::LINE_AA);
        cv::putText(downScaledImg2, decodeInfo1, cv::Point(20, 80), cv::FONT_HERSHEY_SIMPLEX, 0.75, color1, 2, cv::LINE_AA);
        cv::putText(downScaledImg2, decodeInfo2, cv::Point(20, 100), cv::FONT_HERSHEY_SIMPLEX, 0.75, color1, 2, cv::LINE_AA);
        cv::putText(downScaledImg2, decodeInfo3, cv::Point(20, 120), cv::FONT_HERSHEY_SIMPLEX, 0.75, color1, 2, cv::LINE_AA);
        cv::putText(downScaledImg2, decodeMethod, cv::Point(20, 160), cv::FONT_HERSHEY_SIMPLEX, 0.75, color2, 2, cv::LINE_AA);
        cv::putText(downScaledImg2, decodeInfo4, cv::Point(20, 180), cv::FONT_HERSHEY_SIMPLEX, 0.75, color2, 2, cv::LINE_AA);
        cv::putText(downScaledImg2, decodeInfo5, cv::Point(20, 200), cv::FONT_HERSHEY_SIMPLEX, 0.75, color2, 2, cv::LINE_AA);
        cv::putText(downScaledImg2, decodeInfo6, cv::Point(20, 220), cv::FONT_HERSHEY_SIMPLEX, 0.75, color2, 2, cv::LINE_AA);
        cv::putText(downScaledImg2, "Press ESC to exit", cv::Point(20, 980), cv::FONT_HERSHEY_SIMPLEX, 0.75, color3, 2, cv::LINE_AA);
        cv::imshow("Loaded Tiles", downScaledImg2);
        int key = cv::waitKey(1);
        if (key == 27) {
            break;
        }
    }
    cv::destroyAllWindows();
    return 0;
}